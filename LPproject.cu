// Ahmad F. Al Musawi, VCU, 2023

#include <iostream>
#include <fstream>
#include <sstream>

#include <forward_list>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <random>
#include <vector>
#include <set>
#include <list>
#include <cmath>
#include <chrono> 
#include <utility>

double CalculateAUPRC(std::vector<std::pair<float, int>> &data)
{
    // Sort data by prediction score in descending order
    std::sort(data.begin(), data.end(), [](const auto &a, const auto &b)
              { return a.first > b.first; });

    size_t tp = 0, fp = 0, total_positives = 0;
    for (const auto &[score, label] : data)
    {
        if (label == 1)
        {
            ++total_positives;
        }
    }

    double auprc = 0.0, prev_recall = 0.0, prev_precision = 1.0;
    for (const auto &[score, label] : data)
    {
        if (label == 1)
        {
            ++tp;
        }
        else
        {
            ++fp;
        }

        double recall = static_cast<double>(tp) / total_positives;
        double precision = static_cast<double>(tp) / (tp + fp);

        // Calculate area under the curve using trapezoidal rule
        auprc += (recall - prev_recall) * (precision + prev_precision) / 2.0;

        prev_recall = recall;
        prev_precision = precision;
    }

    return auprc;
}

double CalculateAUC(std::vector<std::pair<float, int>> &data)
{
    // Sort data by prediction score in descending order
    std::sort(data.begin(), data.end(), [](const auto &a, const auto &b)
              { return a.first > b.first; });

    size_t tp = 0, fp = 0;
    size_t positive = 0, negative = 0;
    for (const auto &[score, label] : data)
    {
        if (label == 1)
            ++positive;
        else
            ++negative;
    }

    double auc = 0.0, prev_fpr = 0.0, prev_tpr = 0.0;
    for (const auto &[score, label] : data)
    {
        if (label == 1)
            ++tp;
        else
            ++fp;

        double fpr = static_cast<double>(fp) / negative;
        double tpr = static_cast<double>(tp) / positive;

        // Add area of trapezoid
        auc += (fpr - prev_fpr) * (tpr + prev_tpr) / 2.0;

        prev_fpr = fpr;
        prev_tpr = tpr;
    }

    return auc;
}

void shuffleVector(std::vector<std::string> &vec)
{
    // Obtain a time-based seed:
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();

    // Shuffle using std::shuffle
    std::shuffle(vec.begin(), vec.end(), std::default_random_engine(seed));
}

int getIndex(std::vector<int> Nodes, int node)
{
    int temp = -1;
    for (size_t i = 0; i < Nodes.size(); ++i)
    {
        if (Nodes[i] == node)
        {
            temp = i;
            break;
        }
    }
    return temp;
}

__global__ void kernel_AA(bool *A, int *neighbors, int *offsets, float *results, int *Nodes, int numNodes)
{
    // implementing the metric
    // Adamic Adar =  \sum_{z \in N_u \cap N_v} \frac{1}{log K_z}

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    float total = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        int Kz = 0;
                        for (int t = 0; t < numNodes; t++)
                        {
                            if (Nodes[t] == neighbors[i])
                            {
                                Kz = offsets[t + 1] - offsets[t];
                                break;
                            }
                        }
                        total = total + (1 / log10(Kz));
                        break;
                    }
                }
            }
            results[row * numNodes + col] = total;
        }
    }
}
__global__ void kernel_RA(bool *A, int *neighbors, int *offsets, float *results, int *Nodes, int numNodes)
{
    // implementing the metric
    // Resource Allocation =  \sum_{z \in N_u \cap N_v} \frac{1}{K_z}

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    float total = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        int Kz = 0;
                        for (int t = 0; t < numNodes; t++)
                        {
                            if (Nodes[t] == neighbors[i])
                            {
                                Kz = offsets[t + 1] - offsets[t];
                                break;
                            }
                        }
                        total = total + (1 / Kz);
                        break;
                    }
                }
            }
            results[row * numNodes + col] = total;
        }
    }
}

__global__ void kernel_JI(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Jaccard Index =  | N(u) \cap N(v) |/| N(u) \cup N(v) |
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    int numberOfUnion = 0;

    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            numberOfUnion = end1 - start1;
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }

                bool found = false;
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        found = true;
                        break;
                    }
                }
                if (not found)
                {
                    numberOfUnion++;
                }
            }
            results[row * numNodes + col] = numberOfIntersections / numberOfUnion;
        }
    }
}

__global__ void kernel_SI(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Sorensen Index = 2 | N(u) \cap N(v) |/(k_u + k_v)
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }
            }

            // printf("(%d, %d) = %d\n", col, row, numberOfIntersections);
            results[row * numNodes + col] = 2 * numberOfIntersections / ((end1 - start1) + (end2 - start2));
        }
    }
}

__global__ void kernel_CN(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Common Neighbors = | N(u) \cap N(v) |
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }
            }

            // printf("(%d, %d) = %d\n", col, row, numberOfIntersections);
            results[row * numNodes + col] = numberOfIntersections;
        }
    }
}

__global__ void kernel_HPI(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Hub Promoted Index = | N(u) \cap N(v) |/min{k_u, k_v};
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }
            }

            if ((end2 - start2) < (end1 - start1))
            {
                results[row * numNodes + col] = numberOfIntersections / (end2 - start2);
            }
            else
            {
                results[row * numNodes + col] = numberOfIntersections / (end1 - start1);
            }
        }
    }
}

__global__ void kernel_HDI(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Hub depressed Index = | N(u) \cap N(v) |/max{k_u, k_v};
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }
            }

            if ((end2 - start2) > (end1 - start1))
            {
                results[row * numNodes + col] = numberOfIntersections / (end2 - start2);
            }
            else
            {
                results[row * numNodes + col] = numberOfIntersections / (end1 - start1);
            }
        }
    }
}

__global__ void kernel_LLHNI(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Local Leicht-Homle-Newman Index = | N(u) \cap N(v) |/(k_u* k_v);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    int numberOfIntersections = 0;
    if (row < numNodes && col < numNodes)
    {
        // printf("nodes (%d, %d)\n", row, col);
        if (not A[row * numNodes + col])
        {
            for (int i = start1; i < end1; ++i)
            {
                for (int j = start2; j < end2; ++j)
                {
                    if (neighbors[i] == neighbors[j])
                    {
                        numberOfIntersections++;
                        break;
                    }
                }
            }

            results[row * numNodes + col] = numberOfIntersections / ((end2 - start2) * (end1 - start1));
        }
    }
}

__global__ void kernel_PA(bool *A, int *neighbors, int *offsets, float *results, int numNodes)
{
    // implementing the metric
    // Preferential attachment = k_u * k_v
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // indexes of the neighbors of the first node
    int start1 = offsets[col];
    int end1 = offsets[col + 1];

    // indexes of the neighbors of the second node
    int start2 = offsets[row];
    int end2 = offsets[row + 1];

    if (row < numNodes && col < numNodes)
    {

        if (not A[row * numNodes + col])
        {
            results[row * numNodes + col] = (end1 - start1) * (end2 - start2);
        }
    }
}

int main(int argc, char *argv[])
{
    auto start = std::chrono::high_resolution_clock::now();

    bool printing = false;

    // ***********************************   LP in cuda   ***************************
    // STEP ONE: read the edges list, and create the nodes list
    // ******************************************************************************

    // Check if a filename is passed
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <filename>\n";
        return 1;
    }
    // ******************************************************************************
    // number of lines == number of edges

    std::ifstream file(argv[1]); // Use the first argument as the file name
    std::string line;
    int totalEdges = 0;
    int totalNodes = 0;
    std::vector<int> Nodes;
    std::vector<std::string> edges;
    std::vector<std::string> test_edges;
    std::vector<std::string> train_edges;

    if (file.is_open())
    {
        // First pass to count the total number of edges
        while (getline(file, line))
        {
            totalEdges++;
        }

        // Reset the file pointer to the beginning of the file
        file.clear(); // Clear any error flags
        file.seekg(0, std::ios::beg);

        // Second pass to read and process each line

        while (getline(file, line))
        {
            std::istringstream iss(line);
            std::string node1, node2;
            if (!(iss >> node1 >> node2))
            {
                std::cerr << "Error reading line: " << line << '\n';
                continue; // Skip malformed lines
            }

            edges.push_back(line);

            bool found1 = false;
            bool found2 = false;

            // adding nodes
            for (const int &node : Nodes)
            {
                if (node == std::stoi(node1))
                    found1 = true;
                if (node == std::stoi(node2))
                    found2 = true;
                if (found1 && found2)
                    break;
            }

            if (not found1)
            {
                Nodes.push_back(std::stoi(node1));
                totalNodes++;
            }
            if (not found2)
            {
                Nodes.push_back(std::stoi(node2));
                totalNodes++;
            }
        }
        file.close();
    }
    else
    {
        std::cerr << "Unable to open file: " << argv[1] << '\n';
        return 1;
    }

    if (printing)
    {
        for (const int &node : Nodes)
        {
            std::cout << "Node: " << node << std::endl;
        }
        std::cout << "number of Nodes: " << totalNodes << std::endl;
        std::cout << "number of Edges: " << totalEdges << std::endl;
    }

    // ******************************************************************************
    // STEP TWO: generate train and test edges lists
    // ******************************************************************************
    // Shuffle the array
    shuffleVector(edges);

    for (size_t i = 0; i < totalEdges; ++i)
    {
        if (i < totalEdges * 0.3)
        {
            test_edges.push_back(edges[i]);
        }
        else
        {
            train_edges.push_back(edges[i]);
        }
    }

    if (printing)
    {
        std::cout << "Number of elements in train_edges: " << train_edges.size() << std::endl;
        std::cout << "Number of elements in test_edges: " << test_edges.size() << std::endl;
    }

    // ******************************************************************************
    // STEP THREE: creating the node: [neighbors] for the train set
    // ******************************************************************************

    std::vector<std::list<int>> train_adjList(totalNodes);
    std::vector<std::list<int>> test_adjList(totalNodes);

    // first: the train edges
    for (const std::string &edge : train_edges)
    {
        std::istringstream iss(edge);
        std::string v, u;

        iss >> v >> u;

        int i = getIndex(Nodes, std::stoi(v));
        train_adjList[i].push_back(std::stoi(u));
    }

    // second: the test edges
    for (const std::string &edge : test_edges)
    {
        std::istringstream iss(edge);
        std::string v, u;

        iss >> v >> u;

        int i = getIndex(Nodes, std::stoi(v));
        test_adjList[i].push_back(std::stoi(u));
    }

    // ******************************************************************************
    // STEP FOUR: setting the adjacency matrix
    // ******************************************************************************
    // first: for the train set
    bool *A_t = (bool *)malloc(totalNodes * totalNodes * sizeof(bool));
    float *results_t = (float *)malloc(totalNodes * totalNodes * sizeof(float));

    // second: for the test set
    bool *A_p = (bool *)malloc(totalNodes * totalNodes * sizeof(bool));

    // initializing train and test data (A_t, results_t, A_p)
    for (int i = 0; i < totalNodes * totalNodes; i++)
    {
        A_t[i] = false;
        results_t[i] = 0;

        A_p[i] = false;
    };

    // setting the value of the adjacency matrix and results
    for (int i = 0; i < totalNodes - 1; i++)
    {
        for (int j = i + 1; j < totalNodes; j++)
        {
            // for the train set
            bool found = false;
            for (const int &neighbor : train_adjList[i])
            {
                if (Nodes[j] == neighbor)
                {
                    found = true;
                    break;
                }
            }
            if (found)
            {
                A_t[i * totalNodes + j] = true;
                A_t[j * totalNodes + i] = true;

                results_t[i * totalNodes + j] = 999;
                results_t[j * totalNodes + i] = 999;
            }
            // ----------------------------------------
            // for the test set
            found = false;
            for (const int &neighbor : test_adjList[i])
            {
                if (Nodes[j] == neighbor)
                {
                    found = true;
                    break;
                }
            }
            if (found)
            {
                A_p[i * totalNodes + j] = true;
                A_p[j * totalNodes + i] = true;
            }
        }
    }
    // ******************************************************************************
    // Convert (ONLY) the train information to flat array representation to be passed to CUDA
    // ******************************************************************************
    std::vector<int> neighbors, offsets;
    int totalNeighbors = 0;
    for (const auto &list : train_adjList)
    {
        offsets.push_back(totalNeighbors);
        neighbors.insert(neighbors.end(), list.begin(), list.end());
        totalNeighbors += list.size();
    }
    offsets.push_back(totalNeighbors); // Add the end offset

    // Stop timing
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::seconds>(stop - start);
    printf("Data preprocessing = %f\n", duration.count());

    printf("starting Cuda here...\n");
    // ******************************************************************************
    // Initialize time measurement
    float time_difference;
    hipEvent_t startEvent, stopEvent;
    float runtime2, runtime3;

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    // Allocate memory on GPU and copy data
    int *d_neighbors, *d_offsets, *d_Nodes;
    bool *d_A_t;
    float *d_results_t;

    // cuda memory allocation
    hipMalloc(&d_neighbors, neighbors.size() * sizeof(int));
    hipMalloc(&d_offsets, offsets.size() * sizeof(int));
    hipMalloc(&d_A_t, totalNodes * totalNodes * sizeof(bool));
    hipMalloc(&d_results_t, totalNodes * totalNodes * sizeof(float));
    hipMalloc(&d_Nodes, totalNodes * sizeof(int));

    // int *U, *V, *Z;
    // cudaMalloc(&U, totalNodes * sizeof(int));
    // cudaMalloc(&V, totalNodes * sizeof(int));
    // cudaMalloc(&Z, totalNodes * sizeof(int));
    hipEvent_t startEvent2, stopEvent2;
    hipEventCreate(&startEvent2);
    hipEventCreate(&stopEvent2);

    hipEventRecord(startEvent2, 0);

    // cuda memory copying
    hipMemcpy(d_Nodes, Nodes.data(), Nodes.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors.data(), neighbors.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A_t, A_t, totalNodes * totalNodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_results_t, results_t, totalNodes * totalNodes * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(stopEvent2, 0);
    hipEventSynchronize(stopEvent2);
    hipEventElapsedTime(&runtime2, startEvent2, stopEvent2);

    printf("Host to Device memory copying %10.4f ms\n", runtime2);

    // ******************************************************************************
    // Launch the kernel
    dim3 blockDim(8, 8);

    int blockX = (totalNodes + blockDim.x - 1) / blockDim.x;
    int blockY = (totalNodes + blockDim.y - 1) / blockDim.y;

    dim3 gridDim(blockX, blockY);

    // kernel_PA<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    kernel_CN<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    // kernel_HPI<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    // kernel_HDI<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    // kernel_LLHNI<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);

    // kernel_SI<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    // kernel_JI<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, totalNodes);
    // kernel_RA<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, d_Nodes, totalNodes);
    // kernel_AA<<<gridDim, blockDim>>>(d_A_t, d_neighbors, d_offsets, d_results_t, d_Nodes, totalNodes);

    hipEvent_t startEvent3, stopEvent3;
    hipEventCreate(&startEvent3);
    hipEventCreate(&stopEvent3);

    hipEventRecord(startEvent3, 0);
    hipMemcpy(results_t, d_results_t, totalNodes * totalNodes * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stopEvent3, 0);
    hipEventSynchronize(stopEvent3);
    hipEventElapsedTime(&runtime3, startEvent3, stopEvent3);

    printf("Device to Host memory copying %10.4f ms\n", runtime3);

    // Stop time measurement
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time_difference, startEvent, stopEvent);

    printf("Cuda is done here...\n");
    // ************************************************************************************

    std::vector<std::pair<float, int>> data;

    int t = 0;
    int e = 0;
    for (int i = 0; i < totalNodes; i++)
    {
        for (int j = 0; j < totalNodes; j++)
        {
            t = i * totalNodes + j;
            if (not A_t[t])
            {
                e = 0;
                if (A_p[t])
                {
                    e = 1;
                }
                data.push_back({results_t[t], e});
            }
        }
    }

    double auc = CalculateAUC(data);
    // double auc = CalculateAUPRC(data);
    printf("Nodes = %d\nEdges = %d\n", totalNodes, totalEdges);
    printf("%f ms total time.\nAUC was %.2f%\n", time_difference+runtime2+runtime3, auc);

    // Cleanup
    // Free GPU memory
    hipFree(d_neighbors);
    hipFree(d_offsets);
    hipFree(d_A_t);
    hipFree(d_results_t);
    hipFree(d_Nodes);
    // Additionally, if used, free U, V, Z

    // Free CPU memory
    free(A_t);
    free(results_t);
    // Free A_p if it was allocated

    return 0;
}
